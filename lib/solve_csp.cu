#include "hip/hip_runtime.h"
/*
* MIT License
*
* Copyright (c) 2018-2020 char-lie
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/
#include <constraint_graph.hpp>
#include <disparity_graph.hpp>
#include <image.hpp>
#include <indexing_checks.hpp>
#include <indexing.hpp>
#include <labeling_finder.hpp>
#include <lowest_penalties.hpp>
#include <solve_csp.hpp>
#include <types.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#if !defined(__CUDA_ARCH__)
using sp::types::ULONG;
using sp::types::ULONG_ARRAY;
using sp::types::Pixel;
using sp::types::Edge;
using sp::types::Node;
using sp::image::Image;
using sp::graph::disparity::DisparityGraph;
using sp::graph::constraint::ConstraintGraph;
using sp::graph::lowest_penalties::LowestPenalties;
using sp::labeling::finder::choose_best_node;
#endif

__device__ void populate_structures_gpu(
    struct DisparityGraph* disparity_graph,
    struct LowestPenalties* lowest_penalties,
    struct ConstraintGraph* constraint_graph,
    int* nodes_availability,
    unsigned* left_image,
    unsigned* right_image,
    float* min_penalties_pixels,
    float* min_penalties_edges,
    float* reparametrization,
    unsigned height,
    unsigned width,
    unsigned max_value,
    unsigned disparity_levels,
    float threshold,
    float cleanness,
    float smoothness
)
{
    #if defined(__CUDA_ARCH__)
    disparity_graph->left.data = left_image;
    disparity_graph->left.height = height;
    disparity_graph->left.max_value = max_value;
    disparity_graph->left.width = width;

    disparity_graph->right.data = right_image;
    disparity_graph->right.height = height;
    disparity_graph->right.max_value = max_value;
    disparity_graph->right.width = width;

    disparity_graph->cleanness = cleanness;
    disparity_graph->disparity_levels = disparity_levels;
    disparity_graph->reparametrization = reparametrization;
    disparity_graph->smoothness = smoothness;

    lowest_penalties->graph = disparity_graph;
    lowest_penalties->pixels = min_penalties_pixels;
    lowest_penalties->neighborhoods = min_penalties_edges;

    constraint_graph->threshold = threshold;
    constraint_graph->nodes_availability = nodes_availability;
    constraint_graph->lowest_penalties = lowest_penalties;
    constraint_graph->disparity_graph = disparity_graph;
    #endif
}

__device__ void csp_solution_iteration_gpu(
    struct ConstraintGraph* constraint_graph,
    struct Pixel pixel,
    int* changed
)
{
    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    do
    {
        *changed = 0;
        g.sync();
        if (csp_process_pixel(constraint_graph, pixel))
        {
            *changed = 1;
        }
        g.sync();
    } while (*changed);
}

__global__ void csp_iteration_cuda(
    int* nodes_availability,
    int* changed,
    unsigned* left_image,
    unsigned* right_image,
    float* min_penalties_pixels,
    float* min_penalties_edges,
    float* reparametrization,
    unsigned height,
    unsigned width,
    unsigned max_value,
    unsigned disparity_levels,
    float threshold,
    float cleanness,
    float smoothness
)
{
    struct DisparityGraph disparity_graph;
    struct LowestPenalties lowest_penalties;
    struct ConstraintGraph constraint_graph;

    populate_structures_gpu(
        &disparity_graph,
        &lowest_penalties,
        &constraint_graph,
        nodes_availability,
        left_image,
        right_image,
        min_penalties_pixels,
        min_penalties_edges,
        reparametrization,
        height,
        width,
        max_value,
        disparity_levels,
        threshold,
        cleanness,
        smoothness
    );

    struct Pixel pixel;
    pixel.x = blockIdx.x;
    pixel.y = threadIdx.x;

    csp_solution_iteration_gpu(
        &constraint_graph,
        pixel,
        changed
    );
}

__global__ void choose_best_node_gpu(
    int* nodes_availability,
    unsigned* left_image,
    unsigned* right_image,
    float* min_penalties_pixels,
    float* min_penalties_edges,
    float* reparametrization,
    unsigned height,
    unsigned width,
    unsigned max_value,
    unsigned disparity_levels,
    float threshold,
    float cleanness,
    float smoothness,
    unsigned pixel_x,
    unsigned pixel_y
)
{
    struct DisparityGraph disparity_graph;
    struct LowestPenalties lowest_penalties;
    struct ConstraintGraph constraint_graph;

    populate_structures_gpu(
        &disparity_graph,
        &lowest_penalties,
        &constraint_graph,
        nodes_availability,
        left_image,
        right_image,
        min_penalties_pixels,
        min_penalties_edges,
        reparametrization,
        height,
        width,
        max_value,
        disparity_levels,
        threshold,
        cleanness,
        smoothness
    );

    struct Pixel pixel;
    pixel.x = pixel_x;
    pixel.y = pixel_y;
    choose_best_node(&constraint_graph, pixel);
}
