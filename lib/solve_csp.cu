#include <constraint_graph.hpp>
#include <disparity_graph.hpp>
#include <image.hpp>
#include <indexing_checks.hpp>
#include <indexing.hpp>
#include <labeling_finder.hpp>
#include <lowest_penalties.hpp>
#include <solve_csp.hpp>
#include <types.hpp>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>

#if !defined(__CUDA_ARCH__)
using sp::types::ULONG;
using sp::types::ULONG_ARRAY;
using sp::types::Pixel;
using sp::types::Edge;
using sp::types::Node;
using sp::image::Image;
using sp::graph::disparity::DisparityGraph;
using sp::graph::constraint::ConstraintGraph;
using sp::graph::lowest_penalties::LowestPenalties;
using sp::labeling::finder::choose_best_node;
#endif

__device__ void populate_structures_gpu(
    struct DisparityGraph* disparity_graph,
    struct LowestPenalties* lowest_penalties,
    struct ConstraintGraph* constraint_graph,
    int* nodes_availability,
    unsigned* left_image,
    unsigned* right_image,
    float* min_penalties_pixels,
    float* min_penalties_edges,
    float* reparametrization,
    unsigned height,
    unsigned width,
    unsigned max_value,
    unsigned disparity_levels,
    float threshold,
    float cleanness,
    float smoothness
)
{
    #if defined(__CUDA_ARCH__)
    disparity_graph->left.data = left_image;
    disparity_graph->left.height = height;
    disparity_graph->left.max_value = max_value;
    disparity_graph->left.width = width;

    disparity_graph->right.data = right_image;
    disparity_graph->right.height = height;
    disparity_graph->right.max_value = max_value;
    disparity_graph->right.width = width;

    disparity_graph->cleanness = cleanness;
    disparity_graph->disparity_levels = disparity_levels;
    disparity_graph->reparametrization = reparametrization;
    disparity_graph->smoothness = smoothness;

    lowest_penalties->graph = disparity_graph;
    lowest_penalties->pixels = min_penalties_pixels;
    lowest_penalties->neighborhoods = min_penalties_edges;

    constraint_graph->threshold = threshold;
    constraint_graph->nodes_availability = nodes_availability;
    constraint_graph->lowest_penalties = lowest_penalties;
    constraint_graph->disparity_graph = disparity_graph;
    #endif
}

__device__ void csp_solution_iteration_gpu(
    struct ConstraintGraph* constraint_graph,
    struct Pixel pixel,
    int* changed
)
{
    cooperative_groups::grid_group g = cooperative_groups::this_grid();
    do
    {
        *changed = 0;
        g.sync();
        if (csp_process_pixel(constraint_graph, pixel))
        {
            *changed = 1;
        }
        g.sync();
    } while (*changed);
}
