/*
* MIT License
*
* Copyright (c) 2018-2020 char-lie
*
* Permission is hereby granted, free of charge, to any person obtaining a copy
* of this software and associated documentation files (the "Software"), to deal
* in the Software without restriction, including without limitation the rights
* to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
* copies of the Software, and to permit persons to whom the Software is
* furnished to do so, subject to the following conditions:
*
* The above copyright notice and this permission notice shall be included in all
* copies or substantial portions of the Software.
*
* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
* IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
* FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
* AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
* LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
* OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
* SOFTWARE.
*/
#ifndef __CUDA_ARCH__
#include <cuda_csp.hpp>
#include <solve_csp.hpp>

#include <cassert>
#include <cstdio>
#include <vector>

#define cdpErrchk(ans) { cdpAssert((ans), __FILE__, __LINE__); }
__host__ __device__ void cdpAssert(hipError_t code, const char *file, int line)
{
   if (code != hipSuccess)
   {
      printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(code), file, line);
      assert(0);
   }
}

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
   exit(0); \
 }                                                                 \
}

namespace gpu
{

using std::vector;

void prepare_problem(struct ConstraintGraph* graph, struct CUDAProblem* problem)
{
    vector<unsigned> left_image(
        graph->disparity_graph->left.data.begin(),
        graph->disparity_graph->left.data.end()
    );
    vector<float> min_penalties_edges(
        graph->lowest_penalties->neighborhoods.begin(),
        graph->lowest_penalties->neighborhoods.end()
    );
    vector<float> min_penalties_pixels(
        graph->lowest_penalties->pixels.begin(),
        graph->lowest_penalties->pixels.end()
    );
    vector<int> nodes_availability(
        graph->nodes_availability.begin(),
        graph->nodes_availability.end()
    );
    vector<float> reparametrization(
        graph->disparity_graph->reparametrization.begin(),
        graph->disparity_graph->reparametrization.end()
    );
    vector<unsigned> right_image(
        graph->disparity_graph->right.data.begin(),
        graph->disparity_graph->right.data.end()
    );

    cdpErrchk(hipMalloc(
        (void**)&(problem->changed),
        sizeof(int))
    );
    cdpErrchk(hipMalloc(
        &(problem->left_image),
        left_image.size() * sizeof(left_image[0]))
    );
    cdpErrchk(hipMalloc(
        &(problem->min_penalties_edges),
        min_penalties_edges.size() * sizeof(min_penalties_edges[0]))
    );
    cdpErrchk(hipMalloc(
        &(problem->min_penalties_pixels),
        min_penalties_pixels.size() * sizeof(min_penalties_pixels[0]))
    );
    cdpErrchk(hipMalloc(
        &(problem->nodes_availability),
        nodes_availability.size() * sizeof(nodes_availability[0]))
    );
    cdpErrchk(hipMalloc(
        &(problem->reparametrization),
        reparametrization.size() * sizeof(reparametrization[0]))
    );
    cdpErrchk(hipMalloc(
        &(problem->right_image),
        right_image.size() * sizeof(right_image[0]))
    );

    cdpErrchk(hipMemcpy(
        problem->left_image,
        left_image.data(),
        left_image.size() * sizeof(left_image[0]),
        hipMemcpyHostToDevice)
    );
    cdpErrchk(hipMemcpy(
        problem->min_penalties_edges,
        min_penalties_edges.data(),
        min_penalties_edges.size() * sizeof(min_penalties_edges[0]),
        hipMemcpyHostToDevice)
    );
    cdpErrchk(hipMemcpy(
        problem->min_penalties_pixels,
        min_penalties_pixels.data(),
        min_penalties_pixels.size() * sizeof(min_penalties_pixels[0]),
        hipMemcpyHostToDevice)
    );
    cdpErrchk(hipMemcpy(
        problem->nodes_availability,
        nodes_availability.data(),
        nodes_availability.size() * sizeof(nodes_availability[0]),
        hipMemcpyHostToDevice)
    );
    cdpErrchk(hipMemcpy(
        problem->reparametrization,
        reparametrization.data(),
        reparametrization.size() * sizeof(reparametrization[0]),
        hipMemcpyHostToDevice)
    );
    cdpErrchk(hipMemcpy(
        problem->right_image,
        right_image.data(),
        right_image.size() * sizeof(right_image[0]),
        hipMemcpyHostToDevice)
    );
}

void free_problem(struct ConstraintGraph* graph, struct CUDAProblem* problem)
{
    vector<int> nodes_availability(graph->nodes_availability.size());
    cdpErrchk(hipMemcpy(
        nodes_availability.data(),
        problem->nodes_availability,
        nodes_availability.size() * sizeof(nodes_availability[0]),
        hipMemcpyDeviceToHost)
    );

    graph->nodes_availability.assign(
        nodes_availability.begin(),
        nodes_availability.end()
    );

    cdpErrchk(hipDeviceSynchronize());

    cdpErrchk(hipFree(problem->changed));
    cdpErrchk(hipFree(problem->left_image));
    cdpErrchk(hipFree(problem->min_penalties_edges));
    cdpErrchk(hipFree(problem->min_penalties_pixels));
    cdpErrchk(hipFree(problem->nodes_availability));
    cdpErrchk(hipFree(problem->reparametrization));
    cdpErrchk(hipFree(problem->right_image));

    cdpErrchk(hipDeviceSynchronize());
}

void csp_solution_cuda(
    struct ConstraintGraph* graph,
    struct CUDAProblem* problem
)
{
    for (ULONG x = 0; x < graph->disparity_graph->right.width; ++x)
    {
        for (ULONG y = 0; y < graph->disparity_graph->right.height; ++y)
        {
            cudaCheckError();

            choose_best_node_gpu<<<1, 1>>>(

                problem->nodes_availability,

                problem->left_image,
                problem->right_image,
                problem->min_penalties_pixels,
                problem->min_penalties_edges,
                problem->reparametrization,

                graph->disparity_graph->right.height,
                graph->disparity_graph->right.width,
                graph->disparity_graph->right.max_value,
                graph->disparity_graph->disparity_levels,
                graph->threshold,
                graph->disparity_graph->cleanness,
                graph->disparity_graph->smoothness,

                x,
                y
            );

            csp_iteration_cuda<<<
                graph->disparity_graph->right.width,
                graph->disparity_graph->right.height
            >>>(

                problem->nodes_availability,

                problem->changed,
                problem->left_image,
                problem->right_image,
                problem->min_penalties_pixels,
                problem->min_penalties_edges,
                problem->reparametrization,

                graph->disparity_graph->right.height,
                graph->disparity_graph->right.width,
                graph->disparity_graph->right.max_value,
                graph->disparity_graph->disparity_levels,
                graph->threshold,
                graph->disparity_graph->cleanness,
                graph->disparity_graph->smoothness
            );
        }
    }
    cudaCheckError();
    cdpErrchk(hipDeviceSynchronize());
}

}
#endif
